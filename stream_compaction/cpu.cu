#include <cstdio>
#include "cpu.h"

#include "common.h"

namespace StreamCompaction {
    namespace CPU {

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
	        static PerformanceTimer timer;
	        return timer;
        }

		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}

        /**
         * CPU scan (prefix sum).
         * For performance analysis, this is supposed to be a simple for loop.
         * (Optional) For better understanding before starting moving to GPU, you can simulate your GPU scan in this function first.
         */
        void scan(int n, int *odata, const int *idata) {
	        timer().startCpuTimer();

			odata[0] = 0;

			for(int i = 1; i < n; ++i)
			{
				odata[i] = odata[i - 1] + idata[i - 1];
			}

	        timer().endCpuTimer();
        }

        /**
         * CPU stream compaction without using the scan function.
         *
         * @returns the number of elements remaining after compaction.
         */
        int compactWithoutScan(int n, int *odata, const int *idata) {
	        timer().startCpuTimer();

			int outIndex = 0;
			for (int i = 0; i < n; ++i)
			{
				if (idata[i] != 0)
				{
					odata[outIndex++] = idata[i];
				}
			}

	        timer().endCpuTimer();
            return (outIndex);
        }

        /**
         * CPU stream compaction using scan and scatter, like the parallel version.
         *
         * @returns the number of elements remaining after compaction.
         */
        int compactWithScan(int n, int *odata, const int *idata) {

			// Keeping allocations outside of timer
			auto *tempValidator = new int[n];
			auto *scanArray = new int[n];

	        timer().startCpuTimer();

			// 1. Compute temporary array
			for (int i = 0; i < n; ++i)
			{
				tempValidator[i] = (idata[i] != 0 ? 1 : 0);
			}

			// 2. Perform exclusive scan
			scanArray[0] = 0;
			for (int i = 1; i < n; ++i)
			{
				scanArray[i] = scanArray[i - 1] + tempValidator[i - 1];
			}


			// 3. Scatter
			int outIndex = 0;
			for(int i = 0; i < n; ++i)
			{
				if(tempValidator[i] != 0)
				{
					outIndex = scanArray[i];
					odata[outIndex] = idata[i];
				}
			}

	        timer().endCpuTimer();

			delete[] tempValidator;
			delete[] scanArray;

            return (outIndex + 1);
        }
    }
}
