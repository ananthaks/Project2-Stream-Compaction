#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		// Global variables
		int* device_iData;
		int* device_oData;
		int* device_bools;
		int* device_sortedbools;

#define blockSize 128

		void printArray(int n, const int *a, bool abridged = false) {
			
			if("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}


		void PrintCUDAArray(int n, const int* cudaArrayPtr)
		{
			hipDeviceSynchronize();

			int* temp = new int[n];
			hipMemcpy(temp, cudaArrayPtr, n * sizeof(n), hipMemcpyDeviceToHost);

			printArray(n, temp, true);

			delete[] temp;
		}



		/**
		* Kernel to perform a Work efficient scan on a integer array
		*/
		__global__ void kernUpSweep(int n, int two_d, int* outputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			const int two_d_1 = two_d * 2;

			if(index % two_d_1 != 0)
			{
				return;
			}
						
			const int oldIndex = index + two_d - 1;
			const int newIndex = index + two_d_1 - 1;

			const int currData = outputData[newIndex];

			outputData[newIndex] = newIndex != (n - 1) ? currData + outputData[oldIndex] : 0;
		}

		/**
		* Kernel to perform a Work efficient scan on a integer array
		*/
		__global__ void kernDownSweep(int n, int two_d, int* outputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			const int two_d_1 = two_d * 2;

			if (index % two_d_1 != 0)
			{
				return;
			}

			const int oldIndex = index + two_d - 1;
			const int newIndex = index + two_d_1 - 1;

			const int dataAtNewIndex = outputData[newIndex];

			const int t = outputData[oldIndex];
			outputData[oldIndex] = dataAtNewIndex;
			outputData[newIndex] = t + outputData[newIndex];
		}
		
		/**
		* Shifts the whole array to the right by one in parallel
		*/
		__global__ void kernMakeExclusive(int n, int* outputData, int* inputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			outputData[index] = index != 0 ? inputData[index - 1] : 0;
		}


		inline int RoundToPower2(int n)
		{
			int start = 2;

			while(start <= n)
			{
				start *= 2;
			}
			return start;
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
    	{
			const int numElements = RoundToPower2(n);
			const int numTotalBytes = numElements * sizeof(int);
			const int numActualBytes = n * sizeof(int);

			// 1. Allocate the memory in device
			hipMalloc(reinterpret_cast<void**>(&device_oData), numTotalBytes);
			hipMemcpy(device_oData, idata, numActualBytes, hipMemcpyHostToDevice);

			hipDeviceSynchronize();

			timer().startGpuTimer();

			// 2. Compute Block count
			dim3 num_blocks((numElements + blockSize - 1) / blockSize);

			// 3. Call the kernel
			const int log_n = ilog2ceil(numElements);
			// 3a. UpSweep
			int power_2 = 1;
			for(int d = 0; d < log_n; ++d)
			{
				power_2 = (1 << d);
				kernUpSweep << < num_blocks, blockSize >> > (numElements, power_2, device_oData);
			}
			
			// 3b. DownSweep
			for (int d = log_n - 1; d >= 0; --d)
			{
				power_2 = (1 << d);
				kernDownSweep << < num_blocks, blockSize >> > (numElements, power_2, device_oData);
			}

			timer().endGpuTimer();

			hipDeviceSynchronize();
			hipMemcpy(odata, device_oData, numActualBytes, hipMemcpyDeviceToHost);

			// 4. Free up any gpu memory
			hipFree(device_iData);
			hipFree(device_oData);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
    	{
			const int numElements = RoundToPower2(n);
			const int numTotalBytes = numElements * sizeof(int);
			const int numActualBytes = n * sizeof(int);

			int* tempHolder = new int[n];

			printArray(n, idata, true);

			// 1. Allocate the memory in device
			hipMalloc(reinterpret_cast<void**>(&device_iData), numTotalBytes);
			hipMalloc(reinterpret_cast<void**>(&device_oData), numTotalBytes);
			hipMalloc(reinterpret_cast<void**>(&device_bools), numTotalBytes);
			hipMalloc(reinterpret_cast<void**>(&device_sortedbools), numTotalBytes);

			hipMemcpy(device_iData, idata, numActualBytes, hipMemcpyHostToDevice);

			hipDeviceSynchronize();

			timer().startGpuTimer();

			// 2. Compute Block count
			dim3 num_blocks((numElements + blockSize - 1) / blockSize);

			// 3. Call the kernel

			// 3a. Map to bools
			StreamCompaction::Common::kernMapToBoolean << <num_blocks, blockSize >> > (numElements, device_bools, device_iData);
			hipMemcpy(device_sortedbools, device_bools, numActualBytes, hipMemcpyHostToHost);

			// 3b. UpSweep
			const int log_n = ilog2ceil(numElements);
			int power_2 = 1;
			for (int d = 0; d < log_n; ++d)
			{
				power_2 = (1 << d);
				kernUpSweep << < num_blocks, blockSize >> > (numElements, power_2, device_sortedbools);
			}
			
			// 3c. DownSweep
			for (int d = log_n - 1; d >= 0; --d)
			{
				power_2 = (1 << d);
				kernDownSweep << < num_blocks, blockSize >> > (numElements, power_2, device_sortedbools);
			}

			// 3d. Compact
			StreamCompaction::Common::kernScatter << <num_blocks, blockSize >> > (numElements, device_oData, device_iData, device_bools, device_sortedbools);

			// 4. Manually copy from the GPU the bools and check the number of valida values to return
			hipDeviceSynchronize();
			hipMemcpy(tempHolder, device_bools, numActualBytes, hipMemcpyDeviceToHost);
			int count = 0;
			for(int i = 0; i < n; ++i)
			{
				if(tempHolder[i] != 0)
				{
					count++;
				}
			}

			timer().endGpuTimer();

			// Copy over the results
			hipDeviceSynchronize();
			hipMemcpy(odata, device_oData, numActualBytes, hipMemcpyDeviceToHost);

			// 5. Free up any gpu memory
			hipFree(device_iData);
			hipFree(device_oData);
			hipFree(device_bools);
			hipFree(device_sortedbools);
			delete[] tempHolder;

            return (count);
        }
    }
}
