#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		// Global variables
		int* device_iData;
		int* device_oData;

#define blockSize 128


		void printArray(int n, const int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}


		/**
		* Kernel to perform a Work efficient scan on a integer array
		*/
		__global__ void kernUpSweep(int n, int two_d, int* outputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			const int two_d_1 = two_d * 2;

			if(index % two_d_1 != 0)
			{
				return;
			}
						
			const int oldIndex = index + two_d - 1;
			const int newIndex = index + two_d_1 - 1;

			const int currData = outputData[newIndex];

			outputData[newIndex] = newIndex != (n - 1) ? currData + outputData[oldIndex] : 0;
		}

		/**
		* Kernel to perform a Work efficient scan on a integer array
		*/
		__global__ void kernDownSweep(int n, int two_d, int* outputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			const int two_d_1 = two_d * 2;

			if (index % two_d_1 != 0)
			{
				return;
			}

			const int oldIndex = index + two_d - 1;
			const int newIndex = index + two_d_1 - 1;

			const int dataAtNewIndex = outputData[newIndex];

			const int t = outputData[oldIndex];
			outputData[oldIndex] = dataAtNewIndex;
			outputData[newIndex] = t + outputData[newIndex];
		}
		
		/**
		* Shifts the whole array to the right by one in parallel
		*/
		__global__ void kernMakeExclusive(int n, int* outputData, int* inputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			outputData[index] = index != 0 ? inputData[index - 1] : 0;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
    	{
			// 1. Allocate the memory in device
			hipMalloc(reinterpret_cast<void**>(&device_iData), n * sizeof(int));
			hipMalloc(reinterpret_cast<void**>(&device_oData), n * sizeof(int));
			hipMemcpy(device_iData, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipMemcpy(device_oData, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

			timer().startGpuTimer();

			// 2. Compute Block count
			dim3 num_blocks((n + blockSize - 1) / blockSize);

			// 3. Call the kernel
			const int log_n = ilog2ceil(n);
			// 3a. UpSweep
			int power_2 = 1;
			for(int d = 0; d < log_n; ++d)
			{
				power_2 = (1 << d);
				kernUpSweep << < num_blocks, blockSize >> > (n, power_2, device_oData);
			}
			
			// 3b. DownSweep
			for (int d = log_n - 1; d >= 0; --d)
			{
				power_2 = (1 << d);
				kernDownSweep << < num_blocks, blockSize >> > (n, power_2, device_oData);
			}

			timer().endGpuTimer();

			hipDeviceSynchronize();
			hipMemcpy(odata, device_oData, sizeof(int) * n, hipMemcpyDeviceToHost);

			// 4. Free up any gpu memory
			hipFree(device_iData);
			hipFree(device_oData);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
