#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		// Global variables
		int *device_iData;
		int *device_oData;

#define blockSize 512

		void printArray(int n, int *a, bool abridged = false) {
			printf("    [ ");
			for (int i = 0; i < n; i++) {
				if (abridged && i + 2 == 15 && n > 16) {
					i = n - 2;
					printf("... ");
				}
				printf("%3d ", a[i]);
			}
			printf("]\n");
		}

		/**
		 * Kernel to perform a Naive scan on a integer array
		 */
		__global__ void kernScan(int n, int power, int* outputData, int* inputData)
        {
	        const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			// Fetch it once
			const auto curr_data = inputData[index];

			if(index >= power)
			{
				outputData[index] = inputData[index - power] + curr_data;
			}
			else
			{
				outputData[index] = curr_data;
			}
        }

		/**
		 * Shifts the whole array to the right by one in parallel
		 */
		__global__ void kernMakeExclusive(int n, int* outputData, int* inputData)
		{
			const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			outputData[index] = index != 0 ? inputData[index - 1] : 0;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
    	{
			// 1. Allocate the memory in device
			hipMalloc(reinterpret_cast<void**>(&device_iData), n * (sizeof(int)));
			hipMalloc(reinterpret_cast<void**>(&device_oData), n * (sizeof(int)));
			hipMemcpy(device_iData, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

        	timer().startGpuTimer();

			// 2. Compute Block count
			dim3 num_blocks((n + blockSize - 1) / blockSize);
			
			// 3. Call the kernel
			const auto logn = ilog2ceil(n);
			for (auto i = 1; i <= logn; ++i)
			{
				const auto power = 1 << (i - 1);
				kernScan << < num_blocks, blockSize >> > (n, power, device_oData, device_iData);
				
				// Swap
				const auto temp = device_iData;
				device_iData = device_oData;
				device_oData = temp;
			}

			// Make it exclusive as we need that for stream compaction later on
			kernMakeExclusive <<< num_blocks, blockSize >> > (n, device_oData, device_iData);

			hipDeviceSynchronize();
			timer().endGpuTimer();

			hipMemcpy(odata, device_oData, sizeof(int) * n, hipMemcpyDeviceToHost);

			// 4. Free up any gpu memory
			hipFree(device_iData);
			hipFree(device_oData);
        }
    }
}
