#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		// Global variables
		int *device_iData;
		int *device_oData;

#define blockSize 128

		/**
		 * Kernel to perform a Naive scan on a integer array
		 */
		__global__ void kernScan(int n, int power, int* outputData, int* inputData)
        {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			// Fetch it once
			const int currData = inputData[index];

			if(index >= power)
			{
				outputData[index] = inputData[index - power] + currData;
			}
			else
			{
				outputData[index] = currData;
			}
        }

		/**
		 * Shifts the whole array to the right by one in parallel
		 */
		__global__ void kernMakeExclusive(int n, int* outputData, int* inputData)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			outputData[index] = index != 0 ? inputData[index - 1] : 0;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
    	{
			// 1. Allocate the memory in device
			hipMalloc((void**)&device_iData, n * (sizeof(int)));
			hipMalloc((void**)&device_oData, n * (sizeof(int)));
			hipMemcpy(device_iData, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

        	timer().startGpuTimer();

			// 2. Compute Block count
			dim3 numBlocks((n + blockSize - 1) / blockSize);
			
			// 3. Call the kernel
			const int logn = ilog2ceil(n);
			for (int i = 1; i <= logn; ++i)
			{
				const int power = 1 << (i - 1);
				kernScan << < numBlocks, blockSize >> > (n, power, device_oData, device_iData);
				
				// Swap
				int* temp = device_iData;
				device_iData = device_oData;
				device_oData = temp;
			}

			// Make it exclusive as we need that for stream compaction later on
			kernMakeExclusive <<< numBlocks, blockSize >> > (n, device_oData, device_iData);

            timer().endGpuTimer();

			hipDeviceSynchronize();
			hipMemcpy(odata, device_oData, sizeof(int) * n, hipMemcpyDeviceToHost);

			// 4. Free up any gpu memory
			hipFree(device_iData);
			hipFree(device_oData);
        }
    }
}
