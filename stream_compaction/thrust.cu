#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		thrust::device_ptr<int> dev_thrustInputData;
		thrust::device_ptr<int> dev_thrustOutputData;
		int* device_iData;
		int* device_oData;

#define blockSize 512

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
    	{
			const int numTotalBytes = n * sizeof(int);

			hipMalloc(reinterpret_cast<void**>(&device_iData), numTotalBytes);
			hipMalloc(reinterpret_cast<void**>(&device_oData), numTotalBytes);

			hipMemcpy(device_iData, idata, numTotalBytes, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

			dev_thrustInputData = thrust::device_ptr<int>(device_iData);
			dev_thrustOutputData = thrust::device_ptr<int>(device_oData);

			timer().startGpuTimer();
			
			thrust::exclusive_scan(dev_thrustInputData, n + dev_thrustInputData, dev_thrustOutputData);

			timer().endGpuTimer();

			hipDeviceSynchronize();
			hipMemcpy(odata, device_oData, numTotalBytes, hipMemcpyDeviceToHost);

			hipFree(device_iData);
			hipFree(device_oData);
        }
    }
}
